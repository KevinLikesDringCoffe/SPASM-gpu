#include "hip/hip_runtime.h"
#include "../include/spasm_cuda.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

extern "C" void launchSPMVKernel(
    const uint32_t* tilePositions,
    const uint32_t* tileBlockRanges,
    const uint32_t* positionEncodings,
    const float* values,
    const uint16_t* templatePatterns,
    const float* x,
    float* y,
    uint32_t rows,
    uint32_t cols,
    uint32_t tileSize,
    uint32_t numTiles,
    uint32_t maxBlocksPerTile);

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

void allocateSPASMCUDA(const SPASMMatrixHost& host, SPASMMatrixCUDA& cuda) {
    cuda.rows = host.rows;
    cuda.cols = host.cols;
    cuda.tileSize = host.tileSize;
    cuda.numTiles = host.numTiles;
    cuda.numPositions = host.numPositions;
    cuda.numTemplates = host.numTemplates;
    cuda.nnz = host.nnz;

    CUDA_CHECK(hipMalloc(&cuda.d_tilePositions, host.numTiles * 2 * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&cuda.d_tileBlockRanges, host.numTiles * 2 * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&cuda.d_positionEncodings, host.numPositions * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&cuda.d_values, host.nnz * sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda.d_templatePatterns, host.numTemplates * sizeof(uint16_t)));

    CUDA_CHECK(hipMalloc(&cuda.d_x, host.cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda.d_y, host.rows * sizeof(float)));
}

void copySPASMToDevice(const SPASMMatrixHost& host, SPASMMatrixCUDA& cuda) {
    CUDA_CHECK(hipMemcpy(cuda.d_tilePositions, host.tilePositions.data(),
                         host.numTiles * 2 * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda.d_tileBlockRanges, host.tileBlockRanges.data(),
                         host.numTiles * 2 * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda.d_positionEncodings, host.positionEncodings.data(),
                         host.numPositions * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda.d_values, host.values.data(),
                         host.nnz * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda.d_templatePatterns, host.templatePatterns.data(),
                         host.numTemplates * sizeof(uint16_t), hipMemcpyHostToDevice));
}

void freeSPASMCUDA(SPASMMatrixCUDA& cuda) {
    if (cuda.d_tilePositions) hipFree(cuda.d_tilePositions);
    if (cuda.d_tileBlockRanges) hipFree(cuda.d_tileBlockRanges);
    if (cuda.d_positionEncodings) hipFree(cuda.d_positionEncodings);
    if (cuda.d_values) hipFree(cuda.d_values);
    if (cuda.d_templatePatterns) hipFree(cuda.d_templatePatterns);
    if (cuda.d_x) hipFree(cuda.d_x);
    if (cuda.d_y) hipFree(cuda.d_y);
}

void spmvCUDA(const SPASMMatrixCUDA& A, int numIterations) {
    const uint32_t threadsPerBlock = 1024;

    std::vector<uint32_t> hostBlockRanges(A.numTiles * 2);
    CUDA_CHECK(hipMemcpy(hostBlockRanges.data(), A.d_tileBlockRanges,
                         A.numTiles * 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));

    uint32_t maxBlocksPerTile = 0;
    for (uint32_t i = 0; i < A.numTiles; i++) {
        uint32_t numBlocks = hostBlockRanges[i * 2 + 1] - hostBlockRanges[i * 2];
        maxBlocksPerTile = std::max(maxBlocksPerTile, numBlocks);
    }

    std::cout << "Kernel configuration: " << A.numTiles << " blocks, "
              << threadsPerBlock << " threads per block" << std::endl;
    std::cout << "Max 4x4 blocks per tile: " << maxBlocksPerTile << std::endl;

    CUDA_CHECK(hipMemset(A.d_y, 0, A.rows * sizeof(float)));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));

    for (int iter = 0; iter < numIterations; iter++) {
        if (iter > 0) {
            CUDA_CHECK(hipMemset(A.d_y, 0, A.rows * sizeof(float)));
        }

        launchSPMVKernel(
            A.d_tilePositions,
            A.d_tileBlockRanges,
            A.d_positionEncodings,
            A.d_values,
            A.d_templatePatterns,
            A.d_x,
            A.d_y,
            A.rows,
            A.cols,
            A.tileSize,
            A.numTiles,
            threadsPerBlock
        );
    }

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    float avgTime = milliseconds / numIterations;
    double gflops = (2.0 * A.nnz * numIterations) / (milliseconds * 1e6);

    std::cout << "GPU SpMV Performance:" << std::endl;
    std::cout << "  Total time: " << milliseconds << " ms (" << numIterations << " iterations)" << std::endl;
    std::cout << "  Average time: " << avgTime << " ms per iteration" << std::endl;
    std::cout << "  Performance: " << gflops << " GFLOP/s" << std::endl;

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

bool verifyResults(const std::vector<float>& cpu_result,
                  const std::vector<float>& gpu_result,
                  float tolerance) {
    if (cpu_result.size() != gpu_result.size()) {
        std::cerr << "Result size mismatch: CPU=" << cpu_result.size()
                  << ", GPU=" << gpu_result.size() << std::endl;
        return false;
    }

    int errors = 0;
    float maxError = 0.0f;
    int maxErrorIdx = -1;

    for (size_t i = 0; i < cpu_result.size(); i++) {
        float diff = std::abs(cpu_result[i] - gpu_result[i]);
        float relError = (cpu_result[i] != 0.0f) ? diff / std::abs(cpu_result[i]) : diff;

        if (relError > tolerance) {
            if (errors < 10) {
                std::cerr << "Mismatch at index " << i << ": CPU=" << cpu_result[i]
                          << ", GPU=" << gpu_result[i] << ", error=" << relError << std::endl;
            }
            errors++;
        }

        if (diff > maxError) {
            maxError = diff;
            maxErrorIdx = i;
        }
    }

    if (errors > 0) {
        std::cerr << "Total errors: " << errors << " / " << cpu_result.size() << std::endl;
        std::cerr << "Max error: " << maxError << " at index " << maxErrorIdx << std::endl;
        return false;
    }

    std::cout << "Verification PASSED! Max error: " << maxError << std::endl;
    return true;
}
